#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <cutil_inline.h>

// Thread block size
#define BLOCK_SIZE 16

// Matrix dimensions
// (chosen as multiples of the thread block size for simplicity)
#define WA (3 * BLOCK_SIZE) // Matrix A width
#define HA (5 * BLOCK_SIZE) // Matrix A height
#define WB (8 * BLOCK_SIZE) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height

//sequential code implemented on cpu
void computeGold(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j)
		{
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k) 
			{
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
}

// Initialize a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

//Compare the cpu's result with gpu's 
void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) 
  {
    for (i=0; i<width; i++) 
	{
      k = j*width+i;
      if (data1[k] != data2[k]) 
	  {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf(" nTotal Errors = %d n", error_count);
}

/*///////////////////////////////////////////////////////////////////////////////////////////////////// 
__global__ void matrixMul( float* C, float* A, float* B, int wA, int wB)
{
     // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

	// Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) 
	{
        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Bs[k][tx];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}
///////////////////////////////////////////////////////////////////////////////////////////// */

//the function is executed on gpu
__device__  float * GetSubMatrix(float *matrix, int m, int index, int width)
{
	return  matrix+width*BLOCK_SIZE*index+BLOCK_SIZE*m;
}

//Kernel code
__global__ void matrixMul( float* C, float* A, float* B, int wA, int wB)
{
     // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

	// Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;     

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int m= 0; m<wA/BLOCK_SIZE; m++) 
	{
		//get the address of submatrixA
		//float *subA=A+wA*BLOCK_SIZE*by+BLOCK_SIZE*m;
		float *subA=GetSubMatrix(A, m, by, wA);
		//get the address of submatrixB
		//float *subB=B+wB*BLOCK_SIZE*m+BLOCK_SIZE*bx;
		float *subB=GetSubMatrix(B, bx, m, wB);
        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = *(subA+ wA * ty + tx);
        Bs[ty][tx] = *(subB+ wB * ty + tx);

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Bs[k][tx];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    //float *subC = C+wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	float *subC=GetSubMatrix(C, bx, by, wB);
    *(subC + wB * ty + tx)= Csub;
}


int main(int argc, char **argv)
{
	// set seed for rand()
    srand((unsigned)time(NULL));

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) ;
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) ;

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);
    
    // create and start timer
    unsigned int timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);

    // execute the kernel
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
	hipDeviceSynchronize();

    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Processing time: %f (ms) \n", cutGetTimerValue(timer));
    cutilCheckError(cutDeleteTimer(timer));

    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) ;

    // compute reference solution
    float* reference = (float*) malloc(mem_size_C);
    computeGold(reference, h_A, h_B, HA, WA, WB);

    // check result
    CUTBoolean res = cutCompareL2fe(reference, h_C, size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, h_C, WC, HC);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();
	cutilExit(argc, argv);
}

 